#include <iostream>
#include <hip/hip_runtime.h>

__global__ void arrmul(int* a, int* b, int* c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 100) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    const int size = 100;
    int block_num = 10;
    int thread_per_block = 10;

    // Host memory
    int* ha = (int*)malloc(size * sizeof(int));
    int* hb = (int*)malloc(size * sizeof(int));
    int* hc = (int*)malloc(size * sizeof(int));

    // 初始化資料
    for (int i = 0; i < size; i++) {
        ha[i] = rand() % 10;
        hb[i] = rand() % 10;
    }

    // Device memory
    int* da;
    int* db;
    int* dc;
    hipMalloc((void**)&da, size * sizeof(int));
    hipMalloc((void**)&db, size * sizeof(int));
    hipMalloc((void**)&dc, size * sizeof(int));

    // 複製資料到裝置
    hipMemcpy(da, ha, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size * sizeof(int), hipMemcpyHostToDevice);

    // 執行 CUDA kernel
    arrmul<<<block_num, thread_per_block>>>(da, db, dc);
    hipDeviceSynchronize();

    // 複製結果回主機
    hipMemcpy(hc, dc, size * sizeof(int), hipMemcpyDeviceToHost);

    // 印出結果
    for (int i = 0; i < size; i++) {
        printf("%d * %d = %d\n", ha[i], hb[i], hc[i]);
    }

    // 釋放記憶體
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(ha);
    free(hb);
    free(hc);

    return 0;
}
