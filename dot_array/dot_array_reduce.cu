#include <iostream>
#include <hip/hip_runtime.h>

__global__ void dot_array(int *inputa, int *inputb, int *outputc) {
    inputa = inputa + blockDim.x * blockIdx.x;
    inputb = inputb + blockDim.x * blockIdx.x;
    outputc = outputc + blockDim.x * blockIdx.x;
    for(int i = 0;i<blockDim.x;i++){
        outputc[i] = inputa[i] * inputb[i];
    }
    //reduce add
    for(int i=1;i<blockDim.x;i*=2){
        if(threadIdx.x%i==0){
            outputc[threadIdx.x] = outputc[threadIdx.x+i];
        }
    }

}

int main(){
    int* ha = (int*)malloc(20*sizeof(int));
    int* hb = (int*)malloc(20*sizeof(int));
    int* hc = (int*)malloc(20*sizeof(int));

    for (int i = 0; i < 20; i++) {
        ha[i] = rand() % 10;
        hb[i] = rand() % 10;
        printf("a[%2d] = %d\tb[%2d] = %d\n", i, ha[i], i, hb[i]);
    }

    int* da;
    int* db;
    int* dc;

    hipMalloc( (void**)&da, 20*sizeof(int));
    hipMalloc( (void**)&dc, 20*sizeof(int));
    hipMalloc( (void**)&db, 20*sizeof(int));

    hipMemcpy( da, ha, 20*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( db, hb, 20*sizeof(int), hipMemcpyHostToDevice);

    dot_array<<<2,10>>>(da,db,dc);

    hipMemcpy( hc, dc, 20*sizeof(int), hipMemcpyDeviceToHost);

    int res = 0;
    for(int i=0;i<20;i++){
        res = res + hc[i];
    }
    
    printf("Dot product result: %d\n", res);

    hipFree(da );
    hipFree(db );
    hipFree(dc );
    free(ha);
    free(hb);
    free(hc);
}