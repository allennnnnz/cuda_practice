#include <iostream>
#include <hip/hip_runtime.h>

__global__ void dot_array(int *a, int *b, int *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] * b[idx];
}

int main(){
    int* ha = (int*)malloc(20*sizeof(int));
    int* hb = (int*)malloc(20*sizeof(int));
    int* hc = (int*)malloc(20*sizeof(int));

    for (int i = 0; i < 20; i++) {
        ha[i] = rand() % 10;
        hb[i] = rand() % 10;
        printf("a[%2d] = %d\tb[%2d] = %d\n", i, ha[i], i, hb[i]);
    }

    int* da;
    int* db;
    int* dc;

    hipMalloc( (void**)&da, 20*sizeof(int));
    hipMalloc( (void**)&dc, 20*sizeof(int));
    hipMalloc( (void**)&db, 20*sizeof(int));

    hipMemcpy( da, ha, 20*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( db, hb, 20*sizeof(int), hipMemcpyHostToDevice);

    dot_array<<<2,10>>>(da,db,dc);

    hipMemcpy( hc, dc, 20*sizeof(int), hipMemcpyDeviceToHost);

    int res = 0;
    for(int i=0;i<20;i++){
        res = res + hc[i];
    }
    
    printf("Dot product result: %d\n", res);

    hipFree(da );
    hipFree(db );
    hipFree(dc );
    free(ha);
    free(hb);
    free(hc);
}